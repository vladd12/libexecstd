#include <hip/hip_runtime_api.h>
#include <device_info.hpp>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

constexpr int notFound = -1;
constexpr int errorCode = 9999;

__host__ int getDeviceCount()
{
    // Getting device count on computer
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    // If devices not found, check return value
    if (deviceCount == 0) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);
        if (deviceProp.major == errorCode && deviceProp.minor == errorCode)
            deviceCount = notFound;
    }
    return deviceCount;
}

__host__ void printInfo(const hipDeviceProp_t &devProp)
{
    // Выводим иформацию об устройстве
    std::cout << "\n\nDevice name: " << devProp.name << ".\n";
    std::cout << "Compute capability: " << devProp.major << "." << devProp.minor << ".\n";
    std::cout << "Maximum global memory size: " << devProp.totalGlobalMem << " bytes.\n";
    std::cout << "Maximum constant memory size: " << devProp.totalConstMem << " bytes.\n";
    std::cout << "Maximum shared memory size per block: " << devProp.sharedMemPerBlock
              << " bytes.\n";
    std::cout << "Maximum block dimensions: " << devProp.maxThreadsDim[0] << " x "
              << devProp.maxThreadsDim[1] << " x " << devProp.maxThreadsDim[2] << ".\n";
    std::cout << "Maximum grid dimensions: " << devProp.maxGridSize[0] << " x "
              << devProp.maxGridSize[1] << " x " << devProp.maxGridSize[2] << ".\n";
    std::cout << "Max threads per block: " << devProp.maxThreadsPerBlock << ".\n";
    std::cout << "Registers per block: " << devProp.regsPerBlock << " 32-bit registers.\n";
    std::cout << "Warp size: " << devProp.warpSize << ".\n";
    std::cout << "Clock rate: " << devProp.clockRate << " kilohertz.\n";
    std::cout << "Multiprocessor count: " << devProp.multiProcessorCount << "\n\n";
}

void printDeviceInformation()
{
    static const auto devices = getDeviceCount();
    hipDeviceProp_t deviceProp;
    for (auto dev = 0; dev < devices; dev++) {
        hipGetDeviceProperties(&deviceProp, dev);
        printInfo(deviceProp);
    }
}
